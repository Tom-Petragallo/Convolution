#include "hip/hip_runtime.h"
#include <opencv2/opencv.hpp>

#include <vector>
#include <string>
#include <chrono>

using namespace std;

void blur(string in);
void edges(string in);
void id(string in);
void gray_scale(string in);
template <typename T>
void afficher_matrice(vector<vector<T>> M);
template <typename T>
void convolution(vector<vector<T>> M, string in, string out);

int main(int argc, char *argv[])
{

	if (argc != 3) {
		cout << "Utilisation : cmd fichier blur|edges|id|gray" << endl;
		return 0;
	}
	string nom = argv[2];

	if (nom == "blur")
		blur(argv[1]);
	else if (nom == "edges")
		edges(argv[1]);
	else if (nom == "id")
		id(argv[1]);
	else if (nom == "gray")
		gray_scale(argv[1]);
	else {
		cout << "non reconnu" << endl;
		return 0;
	}

	return 0;
}





void id(string in) {
	vector<vector<int>> M = {{0,0,0},{0,1,0},{0,0,0}};
	afficher_matrice<int>(M);
	convolution<int>(M,in,"out_blur.jpg");
}

void blur(string in) {
	vector<vector<float>> M = {{0.0625,0.125,0.0625},{0.125,0.25,0.125},{0.0625,0.125,0.0625}};
	afficher_matrice<float>(M);
	convolution<float>(M,in,"out_blur.jpg");
}

void edges(string in) {
	vector<vector<int>> M = {{-1,-1,-1},{-1,8,-1},{-1,-1,-1}};
	afficher_matrice<int>(M);
	convolution<int>(M,in,"out_edges.jpg");
}

template <typename T>
void afficher_matrice(vector<vector<T>> M) {
	for (int i=0; i<M.size(); i++) {
		for (int j = 0; j < M[i].size(); j++) {
			cout << M[i][j] << ",";
		}
		cout << endl;
	}
}





// Fonction reprise des TP
__global__ void grayscale( unsigned char * rgb, unsigned char * g, std::size_t cols, std::size_t rows ) {
	auto i = blockIdx.x * blockDim.x + threadIdx.x;
	auto j = blockIdx.y * blockDim.y + threadIdx.y;
	if( i < cols && j < rows ) {
		g[ j * cols + i ] = (
									307 * rgb[ 3 * ( j * cols + i ) ]
									+ 604 * rgb[ 3 * ( j * cols + i ) + 1 ]
									+ 113 * rgb[  3 * ( j * cols + i ) + 2 ]
							) / 1024;
	}
}

void gray_scale(string in) {
	cv::Mat m_in = cv::imread(in, cv::IMREAD_UNCHANGED );
	auto rgb = m_in.data;
	auto rows = m_in.rows;
	auto cols = m_in.cols;

	cout << "Taille image : " << rows << " x " << cols << " px" << endl;

	std::vector< unsigned char > g( rows * cols );
	cv::Mat m_out( rows, cols, CV_8UC1, g.data() );
	unsigned char * rgb_d;
	unsigned char * g_d;

	hipError_t status;

	status = hipMalloc( &rgb_d, 3 * rows * cols );
	if (status != hipSuccess) cout << "Erreur malloc rgb_d" << endl;
	status = hipMalloc( &g_d, rows * cols );
	if (status != hipSuccess) cout << "Erreur malloc g_d" << endl;
	status = hipMemcpy( rgb_d, rgb, 3 * rows * cols, hipMemcpyHostToDevice );
	if (status != hipSuccess) cout << "Erreur memcpy HtD rgb_d" << endl;

	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);
	
	dim3 t( 32, 32 );
	dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
	grayscale<<< b, t >>>( rgb_d, g_d, cols, rows );

	auto kernelStatus = hipGetLastError();
	if ( kernelStatus != hipSuccess )
		cout << "CUDA Error : "<< hipGetErrorString(kernelStatus) << " " << endl;

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "Temps d'éxecution : " << elapsedTime << endl;

	status = hipMemcpy( g.data(), g_d, rows * cols, hipMemcpyDeviceToHost );
	if (status != hipSuccess) cout << "Erreur memcpy DtH g_d" << endl;

	cv::imwrite( "out.jpg", m_out );
	hipFree( rgb_d);
	hipFree( g_d);
}





__global__ void convolution_gpu( unsigned char * g, float * M, unsigned char * res, size_t cols, size_t rows ) {
	auto i = blockIdx.x * blockDim.x + threadIdx.x;
	auto j = blockIdx.y * blockDim.y + threadIdx.y;

	if( i < cols && j < rows ) {
		float somme = 0;
		for (int x = -1; x < 2; x++) {
			for (int y = -1; y < 2; y++) {
				int i_c = i + x;
				int j_c = j + y;

				if (i + x < 0)
					i_c = 0;
				if (cols <= i + x)
					i_c = cols - 1;
				if (j + y < 0)
					j_c = 0;
				if (rows <= j + y)
					j_c = rows - 1;

				int i_g = j_c * cols + i_c;
				int gray = g[i_g];
				somme += M[x + 1 + (y + 1)*3] * gray;
			}
		}
		if (somme < 0)
			somme = 0;
		res[i + j * cols] = (unsigned char) somme;
	}
}

template <typename T>
void convolution(vector<vector<T>> M, string in, string out) {
	cv::Mat m_in = cv::imread(in, cv::IMREAD_UNCHANGED );
	unsigned char* g = m_in.data;
	int cols = m_in.cols;
	int rows = m_in.rows;

	cout << cols << "*" << rows << endl;
	vector< unsigned char > res (cols*rows);

	unsigned char * g_d;
	float * M_d;
	unsigned char * res_d;

	hipError_t status;

	status = hipMalloc( &g_d, rows * cols );
	if (status != hipSuccess) cout << "Erreur malloc g_d" << endl;
	status = hipMalloc( &res_d, rows * cols );
	if (status != hipSuccess) cout << "Erreur malloc res_d" << endl;
	status = hipMalloc( &M_d, 9*sizeof(float) );
	if (status != hipSuccess) cout << "Erreur malloc M_d" << endl;

	status = hipMemcpy( g_d, g, rows * cols, hipMemcpyHostToDevice );
	if (status != hipSuccess) cout << "Erreur memcpy HtD g_d" << endl;
	status = hipMemcpy( M_d, g, 9*sizeof(float), hipMemcpyHostToDevice );
	if (status != hipSuccess) cout << "Erreur memcpy HtD M_d" << endl;

	float elapsedTime;
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start,0);

	dim3 t( 32, 32 );
	dim3 b( ( cols - 1) / t.x + 1 , ( rows - 1 ) / t.y + 1 );
	convolution_gpu<<< b, t >>>( g_d, M_d, res_d, cols, rows );

	auto kernelStatus = hipGetLastError();
	if ( kernelStatus != hipSuccess )
		cout << "CUDA Error : "<< hipGetErrorString(kernelStatus) << " " << endl;

	hipEventRecord(stop,0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	cout << "Temps d'éxecution : " << elapsedTime << endl;

	status = hipMemcpy( res.data(), res_d, rows * cols, hipMemcpyDeviceToHost );
	if (status != hipSuccess) cout << "Erreur memcpy DtH res_d" << endl;

	cout << "traitement terminé" << endl;
	cv::Mat m_out( rows, cols, CV_8UC1, res.data() );
	cv::imwrite("out_id.jpg", m_out);
}

